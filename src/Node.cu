#include "Node.cuh"

#include <hip/hip_runtime.h>
#include "Common.cuh"

__host__ Node::Node() : mass(0), particleDensity(0), gx(0), gy(0), u(0), v(0), u2(0), v2(0), ax(0), ay(0), active(false) {
	const int size = numMaterials * sizeof(float);
	gpuErrchk(hipMalloc((void**)&cgx, size));
	gpuErrchk(hipMalloc((void**)&cgy, size));
	gpuErrchk(hipMemset(cgx, 0,  numMaterials * sizeof(float)))
	gpuErrchk(hipMemset(cgy, 0,  numMaterials * sizeof(float)));
}

__host__ Node::~Node()
{
	hipFree(cgx);
	hipFree(cgy);
}
